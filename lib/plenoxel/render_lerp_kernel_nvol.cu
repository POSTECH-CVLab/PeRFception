#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// This is an alternate implementation using the volume rendering formula from Neural Volumes (Lombardi, ToG 2019)
// NOTE: this still uses density as in NeRF, but the key difference is using absolute, instead of relative, transmittance.
// This formula allows for parallel evaluation of points along the ray, since it's just an ablation this is not optimized
// Background is NOT supported
#include <torch/extension.h>
#include "cuda_util.cuh"
#include "data_spec_packed.cuh"
#include "render_util.cuh"

#include <iostream>
#include <cstdint>
#include <tuple>

namespace {
const int WARP_SIZE = 32;

const int TRACE_RAY_CUDA_THREADS = 128;
const int TRACE_RAY_CUDA_RAYS_PER_BLOCK = TRACE_RAY_CUDA_THREADS / WARP_SIZE;

const int TRACE_RAY_BKWD_CUDA_THREADS = 128;
const int TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK = TRACE_RAY_BKWD_CUDA_THREADS / WARP_SIZE;

const int MIN_BLOCKS_PER_SM = 8;
typedef hipcub::WarpReduce<float> WarpReducef;

namespace device {


// * For ray rendering
__device__ __inline__ void trace_ray_nvol(
        const PackedSparseGridSpec& __restrict__ grid,
        SingleRaySpec& __restrict__ ray,
        const RenderOptions& __restrict__ opt,
        uint32_t lane_id,
        float* __restrict__ sphfunc_val,
        WarpReducef::TempStorage& __restrict__ temp_storage,
        float* __restrict__ out) {
    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;

    if (ray.tmin > ray.tmax) {
        out[lane_colorgrp] = (grid.background_nlayers == 0) ? opt.background_brightness : 0.f;
        return;
    }

    float t = ray.tmin;
    float outv = 0.f;

    float total_alpha = 0.f;

    while (t <= ray.tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
            ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
            ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
            ray.pos[j] -= static_cast<float>(ray.l[j]);
        }

        const float skip = compute_skip_dist(ray,
                       grid.links, grid.stride_x,
                       grid.size[2], 0);

        if (skip >= opt.step_size) {
            // For consistency, we skip the by step size
            t += ceilf(skip / opt.step_size) * opt.step_size;
            continue;
        }
        float sigma = trilerp_cuvol_one(
                grid.links, grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                ray.l, ray.pos,
                0);

        if (sigma > opt.sigma_thresh) {
            float lane_color = trilerp_cuvol_one(
                            grid.links,
                            grid.sh_data,
                            grid.stride_x,
                            grid.size[2],
                            grid.sh_data_dim,
                            ray.l, ray.pos, lane_id);
            lane_color *= sphfunc_val[lane_colorgrp_id];

            const float new_total_alpha = fminf(total_alpha + 1.f - _EXP(
                                 -ray.world_step * sigma), 1.f);
            const float weight = new_total_alpha - total_alpha;
            total_alpha = new_total_alpha;

            float lane_color_total = WarpReducef(temp_storage).HeadSegmentedSum(
                                           lane_color, lane_colorgrp_id == 0);
            outv += weight * fmaxf(lane_color_total + 0.5f, 0.f);  // Clamp to [+0, infty)
            if (total_alpha >= 1.f) break;
        }
        t += opt.step_size;
    }

    if (grid.background_nlayers == 0) {
        outv += (1.f - total_alpha) * opt.background_brightness;
    }
    if (lane_colorgrp_id == 0) {
        out[lane_colorgrp] = outv;
    }
}

__device__ __inline__ void trace_ray_nvol_backward(
        const PackedSparseGridSpec& __restrict__ grid,
        const float* __restrict__ grad_output,
        const float* __restrict__ color_cache,
        SingleRaySpec& __restrict__ ray,
        const RenderOptions& __restrict__ opt,
        uint32_t lane_id,
        const float* __restrict__ sphfunc_val,
        float* __restrict__ grad_sphfunc_val,
        WarpReducef::TempStorage& __restrict__ temp_storage,
        float sparsity_loss,
        PackedGridOutputGrads& __restrict__ grads
        ) {
    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;
    const uint32_t leader_mask = 1U | (1U << grid.basis_dim) | (1U << (2 * grid.basis_dim));

    if (ray.tmin > ray.tmax) {
        return;
    }
    float t = ray.tmin;

    const float gout = grad_output[lane_colorgrp];

    float total_alpha = 0.f;
    float last_total_color = 0.f;

    // remat samples
    while (t <= ray.tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
            ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
            ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
            ray.pos[j] -= static_cast<float>(ray.l[j]);
        }
        const float skip = compute_skip_dist(ray,
                       grid.links, grid.stride_x,
                       grid.size[2], 0);
        if (skip >= opt.step_size) {
            // For consistency, we skip the by step size
            t += ceilf(skip / opt.step_size) * opt.step_size;
            continue;
        }

        float sigma = trilerp_cuvol_one(
                grid.links,
                grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                ray.l, ray.pos,
                0);
        if (sigma > opt.sigma_thresh) {
            float lane_color = trilerp_cuvol_one(
                            grid.links,
                            grid.sh_data,
                            grid.stride_x,
                            grid.size[2],
                            grid.sh_data_dim,
                            ray.l, ray.pos, lane_id);
            float weighted_lane_color = lane_color * sphfunc_val[lane_colorgrp_id];

            const float curr_transmit = _EXP(-ray.world_step * sigma);
            const float new_total_alpha = fminf(total_alpha + 1.f - curr_transmit, 1.f);
            const float weight = new_total_alpha - total_alpha;
            bool not_last = new_total_alpha < 1.f;
            total_alpha = new_total_alpha;

            const float lane_color_total = WarpReducef(temp_storage).HeadSegmentedSum(
                                           weighted_lane_color, lane_colorgrp_id == 0) + 0.5f;
            float total_color = fmaxf(lane_color_total, 0.f);
            float color_in_01 = total_color == lane_color_total;
            total_color *= gout; // Clamp to [+0, infty)

            float total_color_c1 = __shfl_sync(leader_mask, total_color, grid.basis_dim);
            total_color += __shfl_sync(leader_mask, total_color, 2 * grid.basis_dim);
            total_color += total_color_c1;

            color_in_01 = __shfl_sync((1U << grid.sh_data_dim) - 1, color_in_01, lane_colorgrp * grid.basis_dim);
            const float grad_common = weight * color_in_01 * gout;
            const float curr_grad_color = sphfunc_val[lane_colorgrp_id] * grad_common;

            if (grid.basis_type != BASIS_TYPE_SH) {
                float curr_grad_sphfunc = lane_color * grad_common;
                const float curr_grad_up2 = __shfl_down_sync((1U << grid.sh_data_dim) - 1,
                        curr_grad_sphfunc, 2 * grid.basis_dim);
                curr_grad_sphfunc += __shfl_down_sync((1U << grid.sh_data_dim) - 1,
                        curr_grad_sphfunc, grid.basis_dim);
                curr_grad_sphfunc += curr_grad_up2;
                if (lane_id < grid.basis_dim) {
                    grad_sphfunc_val[lane_id] += curr_grad_sphfunc;
                }
            }
            trilerp_backward_cuvol_one(grid.links, grads.grad_sh_out,
                    grid.stride_x,
                    grid.size[2],
                    grid.sh_data_dim,
                    ray.l, ray.pos,
                    curr_grad_color, lane_id);

            if (not_last) {
                float curr_grad_sigma =  ray.world_step * curr_transmit * total_color;
                if (sparsity_loss > 0.f) {
                    // Cauchy version (from SNeRG)
                    curr_grad_sigma += sparsity_loss * (4 * sigma / (1 + 2 * (sigma * sigma)));

                    // Alphs version (from PlenOctrees)
                    // curr_grad_sigma += sparsity_loss * _EXP(-pcnt) * ray.world_step;
                }
                if (lane_id == 0) {
                    trilerp_backward_cuvol_one_density(
                            grid.links,
                            grads.grad_density_out,
                            grads.mask_out,
                            grid.stride_x,
                            grid.size[2],
                            ray.l, ray.pos, curr_grad_sigma);
                }
            } else {
                ray.tmax = t;
                last_total_color = total_color;
                break;
            }
        }
        t += opt.step_size;
    }
    if (total_alpha < 1.f) {
        // Never saturatedo
        last_total_color = opt.background_brightness * (
                grad_output[0] + grad_output[1] + grad_output[2]);
    }
    if (last_total_color != 0.f) {
        t = ray.tmin;
        total_alpha = 0.f;

        while (t <= ray.tmax) {
#pragma unroll 3
            for (int j = 0; j < 3; ++j) {
                ray.pos[j] = fmaf(t, ray.dir[j], ray.origin[j]);
                ray.pos[j] = min(max(ray.pos[j], 0.f), grid.size[j] - 1.f);
                ray.l[j] = min(static_cast<int32_t>(ray.pos[j]), grid.size[j] - 2);
                ray.pos[j] -= static_cast<float>(ray.l[j]);
            }
            const float skip = compute_skip_dist(ray,
                    grid.links, grid.stride_x,
                    grid.size[2], 0);
            if (skip >= opt.step_size) {
                // For consistency, we skip the by step size
                t += ceilf(skip / opt.step_size) * opt.step_size;
                continue;
            }
            float sigma = trilerp_cuvol_one(
                    grid.links,
                    grid.density_data,
                    grid.stride_x,
                    grid.size[2],
                    1,
                    ray.l, ray.pos,
                    0);

            const float curr_transmit = _EXP(-ray.world_step * sigma);
            total_alpha = fminf(total_alpha + 1.f - curr_transmit, 1.f);
            // const float weight = new_total_alpha - total_alpha;
            // total_alpha = new_total_alpha;

            if (total_alpha >= 1.f) break;

            float curr_grad_sigma = -ray.world_step * curr_transmit * last_total_color;
            if (lane_id == 0) {
                trilerp_backward_cuvol_one_density(
                        grid.links,
                        grads.grad_density_out,
                        grads.mask_out,
                        grid.stride_x,
                        grid.size[2],
                        ray.l, ray.pos, curr_grad_sigma);
            }

            t += opt.step_size;
        }
    }
}


// BEGIN KERNELS

__launch_bounds__(TRACE_RAY_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void render_ray_kernel(
        PackedSparseGridSpec grid,
        PackedRaysSpec rays,
        RenderOptions opt,
        torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> out) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)) * WARP_SIZE);
    const int ray_id = tid >> 5;
    const int ray_blk_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1F;

    if (lane_id >= grid.sh_data_dim)
        return;

    __shared__ float sphfunc_val[TRACE_RAY_CUDA_RAYS_PER_BLOCK][10];
    __shared__ SingleRaySpec ray_spec[TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    __shared__ typename WarpReducef::TempStorage temp_storage[
        TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    ray_spec[ray_blk_id].set(rays.origins[ray_id].data(),
            rays.dirs[ray_id].data());
    calc_sphfunc(grid, lane_id,
                 ray_id,
                 ray_spec[ray_blk_id].dir,
                 sphfunc_val[ray_blk_id]);
    ray_find_bounds(ray_spec[ray_blk_id], grid, opt, ray_id);
    __syncwarp((1U << grid.sh_data_dim) - 1);

    trace_ray_nvol(
        grid,
        ray_spec[ray_blk_id],
        opt,
        lane_id,
        sphfunc_val[ray_blk_id],
        temp_storage[ray_blk_id],
        out[ray_id].data());
}

__launch_bounds__(TRACE_RAY_BKWD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void render_ray_backward_kernel(
    PackedSparseGridSpec grid,
    const float* __restrict__ grad_output,
    const float* __restrict__ color_cache,
    PackedRaysSpec rays,
    RenderOptions opt,
    bool grad_out_is_rgb,
    float sparsity_loss,
    PackedGridOutputGrads grads) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)) * WARP_SIZE);
    const int ray_id = tid >> 5;
    const int ray_blk_id = threadIdx.x >> 5;
    const int lane_id = threadIdx.x & 0x1F;

    if (lane_id >= grid.sh_data_dim)
        return;

    __shared__ float sphfunc_val[TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK][10];
    __shared__ float grad_sphfunc_val[TRACE_RAY_CUDA_RAYS_PER_BLOCK][10];
    __shared__ SingleRaySpec ray_spec[TRACE_RAY_BKWD_CUDA_RAYS_PER_BLOCK];
    __shared__ typename WarpReducef::TempStorage temp_storage[
        TRACE_RAY_CUDA_RAYS_PER_BLOCK];
    ray_spec[ray_blk_id].set(rays.origins[ray_id].data(),
                             rays.dirs[ray_id].data());
    const float vdir[3] = {ray_spec[ray_blk_id].dir[0],
                     ray_spec[ray_blk_id].dir[1],
                     ray_spec[ray_blk_id].dir[2] };
    if (lane_id < grid.basis_dim) {
        grad_sphfunc_val[ray_blk_id][lane_id] = 0.f;
    }
    calc_sphfunc(grid, lane_id,
                 ray_id,
                 vdir, sphfunc_val[ray_blk_id]);
    if (lane_id == 0) {
        ray_find_bounds(ray_spec[ray_blk_id], grid, opt, ray_id);
    }

    float grad_out[3];
    if (grad_out_is_rgb) {
        const float norm_factor = 2.f / (3 * int(rays.origins.size(0)));
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float resid = color_cache[ray_id * 3 + i] - grad_output[ray_id * 3 + i];
            grad_out[i] = resid * norm_factor;
        }
    } else {
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            grad_out[i] = grad_output[ray_id * 3 + i];
        }
    }

    __syncwarp((1U << grid.sh_data_dim) - 1);
    trace_ray_nvol_backward(
        grid,
        grad_out,
        color_cache + ray_id * 3,
        ray_spec[ray_blk_id],
        opt,
        lane_id,
        sphfunc_val[ray_blk_id],
        grad_sphfunc_val[ray_blk_id],
        temp_storage[ray_blk_id],
        sparsity_loss,
        grads);
    calc_sphfunc_backward(
                 grid, lane_id,
                 ray_id,
                 vdir,
                 sphfunc_val[ray_blk_id],
                 grad_sphfunc_val[ray_blk_id],
                 grads.grad_basis_out);
}

}  // namespace device
}  // namespace

torch::Tensor volume_render_nvol(SparseGridSpec& grid, RaysSpec& rays, RenderOptions& opt) {
    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();


    const auto Q = rays.origins.size(0);

    torch::Tensor results = torch::empty_like(rays.origins);
    const int cuda_n_threads = TRACE_RAY_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, cuda_n_threads);
    device::render_ray_kernel<<<blocks, cuda_n_threads>>>(
            grid, rays, opt,
            // Output
            results.packed_accessor32<float, 2, torch::RestrictPtrTraits>());

    CUDA_CHECK_ERRORS;
    return results;
}

void volume_render_nvol_backward(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor grad_out,
        torch::Tensor color_cache,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    const int cuda_n_threads_render_backward = TRACE_RAY_BKWD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, cuda_n_threads_render_backward);
    device::render_ray_backward_kernel<<<blocks,
        cuda_n_threads_render_backward>>>(
                grid,
                grad_out.data_ptr<float>(),
                color_cache.data_ptr<float>(),
                rays, opt,
                false,
                0.f,
                // Output
                grads);

    CUDA_CHECK_ERRORS;
}

void volume_render_nvol_fused(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor rgb_gt,
        float _,  // not supported
        float sparsity_loss,
        torch::Tensor rgb_out,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    CHECK_INPUT(rgb_gt);
    CHECK_INPUT(rgb_out);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, TRACE_RAY_CUDA_THREADS);
        device::render_ray_kernel<<<blocks, TRACE_RAY_CUDA_THREADS>>>(
                grid, rays, opt,
                // Output
                rgb_out.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
    }

    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q * WARP_SIZE, TRACE_RAY_BKWD_CUDA_THREADS);
        device::render_ray_backward_kernel<<<blocks, TRACE_RAY_BKWD_CUDA_THREADS>>>(
                grid,
                rgb_gt.data_ptr<float>(),
                rgb_out.data_ptr<float>(),
                rays,
                opt,
                true,
                sparsity_loss,
                // Output
                grads);
    }

    CUDA_CHECK_ERRORS;
}
