#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// Loss computation-related kernels

#include <torch/extension.h>
#include <cstdint>
#include <cstdio>
#include "cuda_util.cuh"
#include "render_util.cuh"
#include "data_spec_packed.cuh"

namespace {

const int WARP_SIZE = 32;
const int TV_GRAD_CUDA_THREADS = 256;
const int TV_GRAD_POINTS_PER_BLOCK = TV_GRAD_CUDA_THREADS / WARP_SIZE;
const int MIN_BLOCKS_PER_SM = 4;

typedef hipcub::WarpReduce<float> WarpReducef;

namespace device {

__device__ __inline__
void calculate_ray_scale(float ndc_coeffx,
                         float ndc_coeffy,
                         float z,
                         float maxx,
                         float maxy,
                         float maxz,
                         float* __restrict__ scale) {
    // if (ndc_coeffx > 0.f) {
    //     // FF NDC
    //     scale[0] = maxx * (1.f / 256.f);
    //     scale[1] = maxy * (1.f / 256.f);
    //     scale[2] = maxz * (1.f / 256.f);

        // The following shit does not work
        // // Normalized to [-1, 1] (with 0.5 padding)
        // // const float x_norm = (x + 0.5) / maxx * 2 - 1;
        // // const float y_norm = (y + 0.5) / maxy * 2 - 1;
        // const float z_norm = (z + 0.5) / maxz * 2 - 1;
        //
        // // NDC distances
        // const float disparity = (1 - z_norm) / 2.f; // in [0, 1]
        // scale[0] = (ndc_coeffx * disparity);
        // scale[1] = (ndc_coeffy * disparity);
        // scale[2] = -((z_norm - 1.f + 2.f / maxz) * disparity) / (maxz * 0.5f);
    // } else {
        scale[0] = maxx * (1.f / 256.f);
        scale[1] = maxy * (1.f / 256.f);
        scale[2] = maxz * (1.f / 256.f);
    // }
}


#define CALCULATE_RAY_SCALE(out_name, maxx, maxy, maxz) \
    calculate_ray_scale( \
            ndc_coeffx, ndc_coeffy, \
            z, \
            maxx, \
            maxy, \
            maxz, \
            out_name)

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        float ndc_coeffx, float ndc_coeffy,
        // Output
        float* __restrict__ out) {
    CUDA_GET_THREAD_ID_U64(tid, Q);

    typedef hipcub::BlockReduce<float, 1024> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;
    float scaling[3];
    CALCULATE_RAY_SCALE(scaling, links.size(0), links.size(1), links.size(2));

    const float val000 = (links[x][y][z] >= 0 ?
                          data[links[x][y][z]][idx] : 0.f);
    const float null_val = (ignore_edge ? val000 : 0.f);
    const float val100 = (links[x + 1][y][z] >= 0 ?
                          data[links[x + 1][y][z]][idx] : null_val);
    const float val010 = (links[x][y + 1][z] >= 0 ?
                          data[links[x][y + 1][z]][idx] : null_val);
    const float val001 = (links[x][y][z + 1] >= 0 ?
                          data[links[x][y][z + 1]][idx] : null_val);
    const float dx = (val100 - val000) * scaling[0];
    const float dy = (val010 - val000) * scaling[1];
    const float dz = (val001 - val000) * scaling[2];
    const float tresult = sqrtf(1e-5f + dx * dx + dy * dy + dz * dz);

    const float bresult = BlockReduce(temp_storage).Sum(tresult);
    if (threadIdx.x == 0) {
        atomicAdd(out, bresult * scale);
    }
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_grad_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        float ndc_coeffx, float ndc_coeffy,
        // Output
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    float dummy;
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = tid / (end_dim - start_dim);
    const int z = xyz % (links.size(2) - 1);
    const int xy = xyz / (links.size(2) - 1);
    const int y = xy % (links.size(1) - 1);
    const int x = xy / (links.size(1) - 1);

    if (ignore_edge && links[x][y][z] == 0) return;

    float scaling[3];
    CALCULATE_RAY_SCALE(scaling, links.size(0), links.size(1), links.size(2));

    const float* dptr = data.data();
    const size_t ddim = data.size(1);
    float v000 = 0.f, v100 = 0.f, v010 = 0.f, v001 = 0.f;
    float* gptr000 = &dummy,
         * gptr100 = &dummy,
         * gptr010 = &dummy,
         * gptr001 = &dummy;

    if (links[x][y][z] >= 0) {
        const size_t lnk = links[x][y][z] * ddim + idx;
        v000 = dptr[lnk];
        gptr000 = grad_data + lnk;
    }
    if (links[x + 1][y][z] >= 0) {
        const size_t lnk = links[x + 1][y][z] * ddim + idx;
        v100 = dptr[lnk];
        gptr100 = grad_data + lnk;
    } else if (ignore_edge) v100 = v000;
    if (links[x][y + 1][z] >= 0) {
        const size_t lnk = links[x][y + 1][z] * ddim + idx;
        v010 = dptr[lnk];
        gptr010 = grad_data + lnk;
    } else if (ignore_edge) v010 = v000;
    if (links[x][y][z + 1] >= 0) {
        const size_t lnk = links[x][y][z + 1] * ddim + idx;
        v001 = dptr[lnk];
        gptr001 = grad_data + lnk;
    } else if (ignore_edge) v001 = v000;

    float dx = (v100 - v000);
    float dy = (v010 - v000);
    float dz = (v001 - v000);
    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz);
    dx *= scaling[0];
    dy *= scaling[1];
    dz *= scaling[2];
    if (dx != 0.f) atomicAdd(gptr100, dx * idelta);
    if (dy != 0.f) atomicAdd(gptr010, dy * idelta);
    if (dz != 0.f) atomicAdd(gptr001, dz * idelta);
    atomicAdd(gptr000, -(dx + dy + dz) * idelta);
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void tv_grad_sparse_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> links,
        const torch::PackedTensorAccessor64<float, 2, torch::RestrictPtrTraits> data,
        const int32_t* __restrict__ rand_cells,
        int start_dim, int end_dim,
        float scale,
        size_t Q,
        bool ignore_edge,
        bool ignore_last_z,
        float ndc_coeffx, float ndc_coeffy,
        // Output
        bool* __restrict__ mask_out,
        float* __restrict__ grad_data) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int idx = tid % (end_dim - start_dim) + start_dim;
    const int xyz = rand_cells[tid / (end_dim - start_dim)];
    const int z = xyz % links.size(2);
    const int xy = xyz / links.size(2);
    const int y = xy % links.size(1);
    const int x = xy / links.size(1);

    const int32_t* __restrict__ links_ptr = &links[x][y][z];

    if (ignore_edge && *links_ptr == 0) return;

    float scaling[3];
    CALCULATE_RAY_SCALE(scaling, links.size(0), links.size(1), links.size(2));

    const int offx = links.stride(0), offy = links.stride(1);

    const auto lnk000 = links_ptr[0];
    const auto lnk001 = ((z + 1 < links.size(2)) &&
                         (!ignore_last_z || z != links.size(2) - 2)) ?
                        links_ptr[1] : 0;
    const auto lnk010 = y + 1 < links.size(1) ? links_ptr[offy] : 0;
    const auto lnk100 = x + 1 < links.size(0) ? links_ptr[offx] : 0;
    if (ignore_last_z && z == links.size(2) - 2) return;

    const float v000 = lnk000 >= 0 ? data[lnk000][idx] : 0.f;
    const float null_val = (ignore_edge ? v000 : 0.f);
    const float v001 = lnk001 >= 0 ? data[lnk001][idx] : null_val,
                v010 = lnk010 >= 0 ? data[lnk010][idx] : null_val,
                v100 = lnk100 >= 0 ? data[lnk100][idx] : null_val;

    float dx = (v100 - v000);
    float dy = (v010 - v000);
    float dz = (v001 - v000);
    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz);

    dx *= scaling[0];
    dy *= scaling[1];
    dz *= scaling[2];

#define MAYBE_ADD_SET(lnk, val) if (lnk >= 0 && val != 0.f) { \
    atomicAdd(&grad_data[lnk * data.size(1) + idx], val * idelta); \
    if (mask_out != nullptr) { \
        mask_out[lnk] = true; \
    } \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(lnk000, sm);
    MAYBE_ADD_SET(lnk001, dz);
    MAYBE_ADD_SET(lnk010, dy);
    MAYBE_ADD_SET(lnk100, dx);

#undef MAYBE_ADD_SET
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void msi_tv_grad_sparse_kernel(
        // (reso * 2, reso)
        const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> links,
        // (capacity, n_layers, n_channels)
        const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> msi,
        const int32_t* __restrict__ rand_cells,
        float scale,
        float scale_last,
        size_t Q,
        // Output
        torch::PackedTensorAccessor32<bool, 2, torch::RestrictPtrTraits> msi_mask,
        torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> grad_msi) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int MSI_DATA_DIM = msi.size(2);
    const int channel_id = tid % MSI_DATA_DIM;
    const int msi_idx = rand_cells[tid / MSI_DATA_DIM];

    const int z = msi_idx % msi.size(1);
    int tmp = msi_idx / msi.size(1);

    const int y = tmp % links.size(1);
    const int x = tmp / links.size(1);

    const int nx = (x == links.size(0) - 1) ? 0 : x + 1;
    const int ny = (y == links.size(1) - 1) ? 0 : y + 1;

    const int lnk00 = links[x][y];
    const int lnk01 = links[x][ny];
    const int lnk10 = links[nx][y];

    const float v00 = lnk00 >= 0 ? msi[lnk00][z][channel_id] : 0.f;
    const float v_nxl = (lnk00 >= 0 && z + 1 < msi.size(1)) ? msi[lnk00][z + 1][channel_id] : ((channel_id == MSI_DATA_DIM - 1) ? 0.f : v00);
    const float v01 = lnk01 >= 0 ? msi[lnk01][z][channel_id] : 0.f;
    const float v10 = lnk10 >= 0 ? msi[lnk10][z][channel_id] : 0.f;

    if (channel_id == MSI_DATA_DIM - 1) {
        scale = scale_last;
    }

    float dx = (v10 - v00);
    float dy = (v01 - v00);
    float dz = (v_nxl - v00);
    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz);
    // printf("x=%d y=%d z=%d nx=%d ny=%d dx=%f dy=%f dz=%f scale=%f\n", x, y, z,
    //        nx, ny, dx, dy, dz, scale);

    // const float msi_nlayers = msi.size(1);

    // const float radius = msi_nlayers / (msi_nlayers - z - 0.5f);
    // const float nxl_radius = msi_nlayers / (msi_nlayers - z - 1.5f);
    // const float invr = 1.f / radius;
    // float coord00[3], coord01[3], coord10[3];
    // _equirect2unitvec(x, y, links.size(1), coord00);
    // _equirect2unitvec(x, ny, links.size(1), coord01);
    // _equirect2unitvec(nx, y, links.size(1), coord10);
    // printf("r=%f nlr=%f coord00[%f %f %f] coord01[%f %f %f] coord10[%f %f %f]\n",
    //         radius, nxl_radius,
    //         coord00[0], coord00[1], coord00[2],
    //         coord01[0], coord01[1], coord01[2],
    //         coord10[0], coord10[1], coord10[2]);

    // xsuby3d(coord01, coord00);
    // xsuby3d(coord10, coord00);
    // dx *= _rnorm(coord10) * invr;
    // dy *= _rnorm(coord01) * invr;
    // dz *= 1.f / (nxl_radius - radius);
    dx *= links.size(0) * (1.f / 256.f);
    dy *= links.size(1) * (1.f / 256.f);
    dz *= msi.size(1) * (1.f / 256.f);

#define MAYBE_ADD_SET(link, zz, val) if (link >= 0 && val != 0.f) { \
    atomicAdd(&grad_msi[link][zz][channel_id], val * idelta); \
    if (msi_mask.size(0) > 0) \
        msi_mask[link][zz] = true; \
} \

    const float sm = -(dx + dy + dz);
    MAYBE_ADD_SET(lnk00, z, sm);
    if (z + 1 < msi.size(1)) {
        MAYBE_ADD_SET(lnk00, z + 1, dz);
    }
    MAYBE_ADD_SET(lnk01, z, dy);
    MAYBE_ADD_SET(lnk10, z, dx);
#undef MAYBE_ADD_SET
}

__launch_bounds__(TV_GRAD_CUDA_THREADS, MIN_BLOCKS_PER_SM)
__global__ void lumisphere_tv_grad_sparse_kernel(
        const PackedSparseGridSpec grid,
        const int32_t* __restrict__ rand_cells,
        const float* __restrict__ sphfunc_val,
        const float* __restrict__ sphfunc_val_u,
        float scale,
        size_t Q,
        float ndc_coeffx,
        float ndc_coeffy,
        float dir_factor,
        // Output
        PackedGridOutputGrads grads
        ) {
    CUDA_GET_THREAD_ID_U64(tid, Q);
    const int lane_id = tid & 0x1F;
    if (lane_id >= grid.sh_data_dim) return;
    const int point_id = tid >> 5;
    const int point_blk_id = threadIdx.x >> 5;

    const uint32_t lane_colorgrp_id = lane_id % grid.basis_dim;
    const uint32_t lane_colorgrp = lane_id / grid.basis_dim;

    const int idx = lane_id;

    const int xyz = rand_cells[point_id];
    const int z = xyz % (grid.size[2] - 1);
    const int xy = xyz / (grid.size[2] - 1);
    const int y = xy % (grid.size[1] - 1);
    const int x = xy / (grid.size[1] - 1);

    // __shared__ float grad_sphfunc_val[TV_GRAD_POINTS_PER_BLOCK][10];
    // __shared__ float grad_sphfunc_val_u[TV_GRAD_POINTS_PER_BLOCK][10];
    __shared__ typename WarpReducef::TempStorage temp_storage[TV_GRAD_POINTS_PER_BLOCK];

    uint32_t use_mask = (1U << grid.sh_data_dim) - 1;

    // Currently, will not work for MLP
    __syncwarp(use_mask);

    const int32_t* __restrict__ links_ptr = grid.links +
                         (x * grid.stride_x + y * grid.size[2] + z);

    if (*links_ptr == 0) return;

    float scaling[3];
    CALCULATE_RAY_SCALE(scaling, grid.size[0], grid.size[1], grid.size[2]);

    const int offx = grid.stride_x, offy = grid.size[2];

    const float v000 = links_ptr[0] >= 0 ? grid.sh_data[
                    links_ptr[0] * grid.sh_data_dim + idx] : 0.f;
    const float v001 = links_ptr[1] >= 0 ? grid.sh_data[
                    links_ptr[1] * grid.sh_data_dim + idx] : v000,
                v010 = links_ptr[offy] >= 0 ? grid.sh_data[
                    links_ptr[offy] * grid.sh_data_dim + idx] : v000,
                v100 = links_ptr[offx] >= 0 ? grid.sh_data[
                    links_ptr[offx] * grid.sh_data_dim + idx] : v000;

    const float sv = sphfunc_val[lane_colorgrp_id];
    const float v000a = v000 * sv,
                v001a = v001 * sv,
                v010a = v010 * sv,
                v100a = v100 * sv;
    const float v000u = v000 * sphfunc_val_u[lane_colorgrp_id];

    const bool is_leader = lane_colorgrp_id == 0;
    float v000a_sum = WarpReducef(temp_storage[point_blk_id]).HeadSegmentedSum(
                            v000a, is_leader);
    float v001a_sum = WarpReducef(temp_storage[point_blk_id]).HeadSegmentedSum(
                            v001a, is_leader);
    float v010a_sum = WarpReducef(temp_storage[point_blk_id]).HeadSegmentedSum(
                            v010a, is_leader);
    float v100a_sum = WarpReducef(temp_storage[point_blk_id]).HeadSegmentedSum(
                            v100a, is_leader);
    float v000u_sum = WarpReducef(temp_storage[point_blk_id]).HeadSegmentedSum(
                            v000u, is_leader);

    const float scale_u = dir_factor;

    float dx = (v100a_sum - v000a_sum) * scaling[0];
    float dy = (v010a_sum - v000a_sum) * scaling[1];
    float dz = (v001a_sum - v000a_sum) * scaling[2];
    float du = (v000u_sum - v000a_sum) * scale_u;

    int leader_id = lane_colorgrp * grid.basis_dim;
    dx = __shfl_sync(use_mask, dx, leader_id);
    dy = __shfl_sync(use_mask, dy, leader_id);
    dz = __shfl_sync(use_mask, dz, leader_id);
    du = __shfl_sync(use_mask, du, leader_id);

    const float idelta = scale * rsqrtf(1e-9f + dx * dx + dy * dy + dz * dz + du * du);

    dx *= scaling[0];
    dy *= scaling[1];
    dz *= scaling[2];
    du *= scale_u;

#define MAYBE_ADD_SET(gp, val) if (links_ptr[gp] >= 0 && val != 0.f) { \
    atomicAdd(&grads.grad_sh_out[links_ptr[gp] * grid.sh_data_dim + idx], val * idelta); \
    if (grads.mask_out != nullptr) { \
        grads.mask_out[links_ptr[gp]] = true; \
    } \
} \

    const float sm = -dx * sv - dy * sv - dz * sv +
                      du * (sphfunc_val_u[lane_colorgrp_id] - sv);
    MAYBE_ADD_SET(0, sm);
    MAYBE_ADD_SET(1, dz * sv);
    MAYBE_ADD_SET(offy, dy * sv);
    MAYBE_ADD_SET(offx, dx * sv);

#undef MAYBE_ADD_SET

    // TODO
    // __syncwarp(use_mask);
    // if (lane_id < grid.basis_dim) {
    //     calc_sphfunc_backward(
    //             grid,
    //             lane_id,
    //             point_id,
    //             dir,
    //             sphfunc_val[point_blk_id],
    //             grad_sphfunc_val_v[point_blk_id],
    //             grad_basis_out);
    //     calc_sphfunc_backward(
    //             grid,
    //             lane_id,
    //             point_id,
    //             dir_u,
    //             sphfunc_val_u[point_blk_id],
    //             grad_sphfunc_val[point_blk_id],
    //             grad_basis_out);
    //     calc_sphfunc_backward(
    //             grid,
    //             lane_id,
    //             point_id,
    //             dir_v,
    //             sphfunc_val_v[point_blk_id],
    //             grad_sphfunc_val_v[point_blk_id],
    //             grad_basis_out);
    // }
}

}  // namespace device
}  // namespace


torch::Tensor tv(torch::Tensor links, torch::Tensor data,
                 int start_dim, int end_dim,
                 bool use_logalpha,
                 float logalpha_delta,
                 bool ignore_edge,
                 float ndc_coeffx,
                 float ndc_coeffy) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);

    int nl = (links.size(0) - 1) * (links.size(1) - 1) * (links.size(2) - 1);
    size_t Q = nl * size_t(end_dim - start_dim);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, TV_GRAD_CUDA_THREADS);
    torch::Tensor result = torch::zeros({}, data.options());
    device::tv_kernel<<<blocks, TV_GRAD_CUDA_THREADS>>>(
            links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            start_dim,
            end_dim,
            1.f / nl,
            Q,
            ignore_edge,
            ndc_coeffx, ndc_coeffy,
            // Output
            result.data_ptr<float>());
    CUDA_CHECK_ERRORS;
    return result;
}

void tv_grad(torch::Tensor links,
             torch::Tensor data,
             int start_dim, int end_dim,
             float scale,
             bool use_logalpha,
             float logalpha_delta,
             bool ignore_edge,
             float ndc_coeffx,
             float ndc_coeffy,
             torch::Tensor grad_data) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    CHECK_INPUT(grad_data);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(grad_data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);
    TORCH_CHECK(grad_data.ndimension() == 2);

    int nl = (links.size(0) - 1) * (links.size(1) - 1) * (links.size(2) - 1);
    size_t Q = nl * size_t(end_dim - start_dim);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    device::tv_grad_kernel<<<blocks, cuda_n_threads>>>(
            links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            start_dim,
            end_dim,
            scale / nl,
            Q,
            ignore_edge,
            ndc_coeffx, ndc_coeffy,
            // Output
            grad_data.data_ptr<float>());
    CUDA_CHECK_ERRORS;
}

void tv_grad_sparse(torch::Tensor links,
             torch::Tensor data,
             torch::Tensor rand_cells,
             torch::Tensor mask_out,
             int start_dim, int end_dim,
             float scale,
             bool use_logalpha,
             float logalpha_delta,
             bool ignore_edge,
             bool ignore_last_z,
             float ndc_coeffx,
             float ndc_coeffy,
             torch::Tensor grad_data) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(links);
    CHECK_INPUT(grad_data);
    CHECK_INPUT(rand_cells);
    CHECK_INPUT(mask_out);
    TORCH_CHECK(data.is_floating_point());
    TORCH_CHECK(grad_data.is_floating_point());
    TORCH_CHECK(!links.is_floating_point());
    TORCH_CHECK(data.ndimension() == 2);
    TORCH_CHECK(links.ndimension() == 3);
    TORCH_CHECK(grad_data.ndimension() == 2);

    int nl = rand_cells.size(0);
    size_t Q = rand_cells.size(0) * size_t(end_dim - start_dim);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    device::tv_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
            links.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            data.packed_accessor64<float, 2, torch::RestrictPtrTraits>(),
            rand_cells.data_ptr<int32_t>(),
            start_dim,
            end_dim,
            scale / nl,
            Q,
            ignore_edge,
            ignore_last_z,
            ndc_coeffx, ndc_coeffy,
            // Output
            (mask_out.dim() > 0) ? mask_out.data_ptr<bool>() : nullptr,
            grad_data.data_ptr<float>());
    CUDA_CHECK_ERRORS;
}

void msi_tv_grad_sparse(
             // (reso * 2, reso)
             torch::Tensor links,
             // (capacity, n_layers, n_channels)
             torch::Tensor msi,
             torch::Tensor rand_cells,
             torch::Tensor mask_out,
             float scale,
             float scale_last,
             torch::Tensor grad_msi) {
    DEVICE_GUARD(msi);
    CHECK_INPUT(links);
    CHECK_INPUT(msi);
    CHECK_INPUT(grad_msi);
    CHECK_INPUT(rand_cells);
    CHECK_INPUT(mask_out);
    TORCH_CHECK(msi.is_floating_point());
    TORCH_CHECK(grad_msi.is_floating_point());

    int nl = rand_cells.size(0);
    size_t Q = rand_cells.size(0) * msi.size(2);

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    device::msi_tv_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
            links.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
            msi.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
            rand_cells.data_ptr<int32_t>(),
            scale / nl,
            scale_last / nl,
            Q,
            // Output
            mask_out.packed_accessor32<bool, 2, torch::RestrictPtrTraits>(),
            grad_msi.packed_accessor32<float, 3, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
}

void lumisphere_tv_grad_sparse(
             SparseGridSpec& grid,
             torch::Tensor rand_cells,
             torch::Tensor basis_fn,
             torch::Tensor basis_fn_u,
             float scale,
             float ndc_coeffx,
             float ndc_coeffy,
             float dir_factor,
             GridOutputGrads& grads) {
    DEVICE_GUARD(grid.sh_data);
    CHECK_INPUT(rand_cells);
    CHECK_INPUT(basis_fn);
    CHECK_INPUT(basis_fn_u);
    TORCH_CHECK(basis_fn.ndimension() == 1);
    grid.check();
    grads.check();

    int nl = rand_cells.size(0);
    size_t Q = rand_cells.size(0) * WARP_SIZE;

    const int cuda_n_threads = TV_GRAD_CUDA_THREADS;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    device::lumisphere_tv_grad_sparse_kernel<<<blocks, cuda_n_threads>>>(
            grid,
            rand_cells.data_ptr<int32_t>(),
            basis_fn.data_ptr<float>(),
            basis_fn_u.data_ptr<float>(),
            scale / nl,
            Q,
            ndc_coeffx, ndc_coeffy,
            dir_factor,
            // Output
            grads);
    CUDA_CHECK_ERRORS;
}
