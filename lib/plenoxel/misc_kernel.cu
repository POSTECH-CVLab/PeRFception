#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// Miscellaneous kernels (3D mask dilate, weight thresholding)

#include <torch/extension.h>
#include <cstdint>
#include <cstdio>
#include <iostream>
#include "cuda_util.cuh"
#include "render_util.cuh"
#include "data_spec_packed.cuh"
#include "cubemap_util.cuh"

namespace {

const int MISC_CUDA_THREADS = 256;
const int MISC_MIN_BLOCKS_PER_SM = 4;
namespace device {

// Can also implement using convs.....
__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void dilate_kernel(
        const torch::PackedTensorAccessor32<bool, 3, torch::RestrictPtrTraits> grid,
        // Output
        torch::PackedTensorAccessor32<bool, 3, torch::RestrictPtrTraits> out_grid) {
    CUDA_GET_THREAD_ID(tid, grid.size(0) * grid.size(1) * grid.size(2));

    const int z = tid % grid.size(2);
    const int xy = tid / grid.size(2);
    const int y = xy % grid.size(1);
    const int x = xy / grid.size(1);

    int xl = max(x - 1, 0), xr = min(x + 1, (int) grid.size(0) - 1);
    int yl = max(y - 1, 0), yr = min(y + 1, (int) grid.size(1) - 1);
    int zl = max(z - 1, 0), zr = min(z + 1, (int) grid.size(2) - 1);

    out_grid[x][y][z] =
        grid[xl][yl][zl] | grid[xl][yl][z] | grid[xl][yl][zr] |
        grid[xl][y][zl] | grid[xl][y][z] | grid[xl][y][zr] |
        grid[xl][yr][zl] | grid[xl][yr][z] | grid[xl][yr][zr] |

        grid[x][yl][zl] | grid[x][yl][z] | grid[x][yl][zr] |
        grid[x][y][zl] | grid[x][y][z] | grid[x][y][zr] |
        grid[x][yr][zl] | grid[x][yr][z] | grid[x][yr][zr] |

        grid[xr][yl][zl] | grid[xr][yl][z] | grid[xr][yl][zr] |
        grid[xr][y][zl] | grid[xr][y][z] | grid[xr][y][zr] |
        grid[xr][yr][zl] | grid[xr][yr][z] | grid[xr][yr][zr];
}

// Probably can speed up the following functions, however they are really not
// the bottleneck

// ** Distance transforms
// TODO: Maybe replace this with an euclidean distance transform eg PBA
// Actual L-infty distance transform; turns out this is slower than the geometric way
__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void accel_linf_dist_transform_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> grid,
        int32_t* __restrict__ tmp,
        int d2) {
    const int d0 = d2 == 0 ? 1 : 0;
    const int d1 = 3 - d0 - d2;
    CUDA_GET_THREAD_ID(tid, grid.size(d0) * grid.size(d1));
    int32_t* __restrict__ tmp_ptr = tmp + tid * grid.size(d2);
    int l[3];
    l[d0] = tid / grid.size(1);
    l[d1] = tid % grid.size(1);
    l[d2] = 0;
    const int INF = 0x3f3f3f3f;
    int d01_dist = INF;
    int d2_dist = INF;
    for (; l[d2] < grid.size(d2); ++l[d2]) {
        const int val = grid[l[0]][l[1]][l[2]];
        int32_t cdist = max(- val, 0);
        if (d2 == 0 && cdist)
            cdist = INF;
        const int32_t pdist = max(d2_dist, d01_dist);

        if (cdist < pdist) {
            d01_dist = cdist;
            d2_dist = 0;
        }
        tmp_ptr[l[d2]] = min(cdist, pdist);
        ++d2_dist;
    }

    l[d2] = grid.size(d2) - 1;
    d01_dist = INF;
    d2_dist = INF;
    for (; l[d2] >= 0; --l[d2]) {
        const int val = grid[l[0]][l[1]][l[2]];
        int32_t cdist = max(- val, 0);
        if (d2 == 0 && cdist)
            cdist = INF;
        const int32_t pdist = max(d2_dist, d01_dist);

        if (cdist < pdist) {
            d01_dist = cdist;
            d2_dist = 0;
        }
        if (cdist) {
            grid[l[0]][l[1]][l[2]] = -min(tmp_ptr[l[d2]], min(cdist, pdist));
        }
        ++d2_dist;
    }
}

// Geometric L-infty distance transform-ish thing
__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void accel_dist_set_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> grid,
        bool* __restrict__ tmp) {
    int sz_x = grid.size(0);
    int sz_y = grid.size(1);
    int sz_z = grid.size(2);
    CUDA_GET_THREAD_ID(tid, sz_x * sz_y * sz_z);

    int z = tid % grid.size(2);
    const int xy = tid / grid.size(2);
    int y = xy % grid.size(1);
    int x = xy / grid.size(1);

    bool* tmp_base = tmp;

    if (grid[x][y][z] >= 0) {
        while (sz_x > 1 && sz_y > 1 && sz_z > 1) {
            // Propagate occupied cell throughout the temp tree parent nodes
            x >>= 1;
            y >>= 1;
            z >>= 1;
            sz_x = int_div2_ceil(sz_x);
            sz_y = int_div2_ceil(sz_y);
            sz_z = int_div2_ceil(sz_z);

            const int idx = x * sz_y * sz_z + y * sz_z + z;
            // printf("s %d  %d %d %d  %d\n", tid, x, y, z, idx);
            tmp_base[idx] = true;
            tmp_base += sz_x * sz_y * sz_z;
        }
    }
}

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void accel_dist_prop_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> grid,
        const bool* __restrict__ tmp) {
    int sz_x = grid.size(0);
    int sz_y = grid.size(1);
    int sz_z = grid.size(2);
    CUDA_GET_THREAD_ID(tid, sz_x * sz_y * sz_z);

    int z = tid % grid.size(2);
    const int xy = tid / grid.size(2);
    int y = xy % grid.size(1);
    int x = xy / grid.size(1);
    const bool* tmp_base = tmp;
    int32_t* __restrict__ val = &grid[x][y][z];

    if (*val < 0) {
        int result = -1;
        while (sz_x > 1 && sz_y > 1 && sz_z > 1) {
            // Find the lowest set parent cell if it exists
            x >>= 1;
            y >>= 1;
            z >>= 1;
            sz_x = int_div2_ceil(sz_x);
            sz_y = int_div2_ceil(sz_y);
            sz_z = int_div2_ceil(sz_z);

            const int idx = x * sz_y * sz_z + y * sz_z + z;
            // printf("p %d  %d %d %d  %d tb[%d] , %d %d %d\n", tid, x, y, z, idx, tmp_base[idx],
            //         sz_x, sz_y, sz_z);
            if (tmp_base[idx]) {
                break;
            }
            result -= 1;
            tmp_base += sz_x * sz_y * sz_z;
        }
        *val = result;
    }
}

// Fast single-channel rendering for weight-thresholding
__device__ __inline__ void grid_trace_ray(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        data,
        SingleRaySpec ray,
        const float* __restrict__ offset,
        const float* __restrict__ scaling,
        float step_size,
        float stop_thresh,
        bool last_sample_opaque,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {

    // Warning: modifies ray.origin
    transform_coord(ray.origin, scaling, offset);
    // Warning: modifies ray.dir
    const float world_step = _get_delta_scale(scaling, ray.dir) * step_size;

    float t, tmax;
    {
        float t1, t2;
        t = 0.0f;
        tmax = 2e3f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float invdir = 1.0 / ray.dir[i];
            t1 = (-0.5f - ray.origin[i]) * invdir;
            t2 = (data.size(i) - 0.5f  - ray.origin[i]) * invdir;
            if (ray.dir[i] != 0.f) {
                t = max(t, min(t1, t2));
                tmax = min(tmax, max(t1, t2));
            }
        }
    }

    if (t > tmax) {
        // Ray doesn't hit box
        return;
    }
    float pos[3];
    int32_t l[3];

    float log_light_intensity = 0.f;
    const int stride0 = data.size(1) * data.size(2);
    const int stride1 = data.size(2);
    while (t <= tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            pos[j] = ray.origin[j] + t * ray.dir[j];
            pos[j] = min(max(pos[j], 0.f), data.size(j) - 1.f);
            l[j] = (int32_t) pos[j];
            l[j] = min(l[j], data.size(j) - 2);
            pos[j] -= l[j];
        }

        float log_att;
        const int idx = l[0] * stride0 + l[1] * stride1 + l[2];

        float sigma;
        {
            // Trilerp
            const float* __restrict__ sigma000 = data.data() + idx;
            const float* __restrict__ sigma100 = sigma000 + stride0;
            const float ix0y0 = lerp(sigma000[0], sigma000[1], pos[2]);
            const float ix0y1 = lerp(sigma000[stride1], sigma000[stride1 + 1], pos[2]);
            const float ix1y0 = lerp(sigma100[0], sigma100[1], pos[2]);
            const float ix1y1 = lerp(sigma100[stride1], sigma100[stride1 + 1], pos[2]);
            const float ix0 = lerp(ix0y0, ix0y1, pos[1]);
            const float ix1 = lerp(ix1y0, ix1y1, pos[1]);
            sigma = lerp(ix0, ix1, pos[0]);
        }
        if (last_sample_opaque && t + step_size > tmax) {
            sigma += 1e9;
            log_light_intensity = 0.f;
        }

        if (sigma > 1e-8f) {
            log_att = -world_step * sigma;
            const float weight = _EXP(log_light_intensity) * (1.f - _EXP(log_att));
            log_light_intensity += log_att;
            float* __restrict__ max_wt_ptr_000 = grid_weight.data() + idx;
            atomicMax(max_wt_ptr_000, weight);
            atomicMax(max_wt_ptr_000 + 1, weight);
            atomicMax(max_wt_ptr_000 + stride1, weight);
            atomicMax(max_wt_ptr_000 + stride1 + 1, weight);
            float* __restrict__ max_wt_ptr_100 = max_wt_ptr_000 + stride0;
            atomicMax(max_wt_ptr_100, weight);
            atomicMax(max_wt_ptr_100 + 1, weight);
            atomicMax(max_wt_ptr_100 + stride1, weight);
            atomicMax(max_wt_ptr_100 + stride1 + 1, weight);

            if (_EXP(log_light_intensity) < stop_thresh) {
                break;
            }
        }
        t += step_size;
    }
}

// __global__ void sample_cubemap_kernel(
//     const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits>
//         cubemap,
//     const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits>
//         dirs,
//     int Q,
//     bool eac,
//     // Output
//     torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits>
//         result) {
//     CUDA_GET_THREAD_ID(tid, Q);
//
//     const int chnl_id = tid % cubemap.size(3);
//     const int ray_id = tid / cubemap.size(3);
//
//     const int face_reso = cubemap.size(1);
//
//     CubemapCoord coord = dir_to_cubemap_coord(dirs[ray_id].data(), face_reso, eac);
//     CubemapBilerpQuery query = cubemap_build_query(coord, face_reso);
//     result[ray_id][chnl_id] = cubemap_sample(
//             cubemap.data(),
//             query,
//             face_reso,
//             cubemap.size(3),
//             chnl_id);
// }

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void grid_weight_render_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        data,
    PackedCameraSpec cam,
    float step_size,
    float stop_thresh,
    bool last_sample_opaque,
    const float* __restrict__ offset,
    const float* __restrict__ scaling,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    float dir[3], origin[3];
    cam2world_ray(ix, iy, cam, dir, origin);
    grid_trace_ray(
        data,
        SingleRaySpec(origin, dir),
        offset,
        scaling,
        step_size,
        stop_thresh,
        last_sample_opaque,
        grid_weight);
}

}  // namespace device
}  // namespace

torch::Tensor dilate(torch::Tensor grid) {
    DEVICE_GUARD(grid);
    CHECK_INPUT(grid);
    TORCH_CHECK(!grid.is_floating_point());
    TORCH_CHECK(grid.ndimension() == 3);

    int Q = grid.size(0) * grid.size(1) * grid.size(2);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);
    torch::Tensor result = torch::empty_like(grid);
    device::dilate_kernel<<<blocks, MISC_CUDA_THREADS>>>(
            grid.packed_accessor32<bool, 3, torch::RestrictPtrTraits>(),
            // Output
            result.packed_accessor32<bool, 3, torch::RestrictPtrTraits>());
    return result;
}

void accel_dist_prop(torch::Tensor grid) {
    // Grid here is links array from the sparse grid
    DEVICE_GUARD(grid);
    CHECK_INPUT(grid);
    TORCH_CHECK(!grid.is_floating_point());
    TORCH_CHECK(grid.ndimension() == 3);

    int64_t sz_x = grid.size(0);
    int64_t sz_y = grid.size(1);
    int64_t sz_z = grid.size(2);

    int Q = grid.size(0) * grid.size(1) * grid.size(2);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    int64_t req_size = 0;
    while (sz_x > 1 && sz_y > 1 && sz_z > 1) {
        sz_x = int_div2_ceil(sz_x);
        sz_y = int_div2_ceil(sz_y);
        sz_z = int_div2_ceil(sz_z);
        req_size += sz_x * sz_y * sz_z;
    }

    auto tmp_options = torch::TensorOptions()
                  .dtype(torch::kBool)
                  .layout(torch::kStrided)
                  .device(grid.device())
                  .requires_grad(false);
    torch::Tensor tmp = torch::zeros({req_size}, tmp_options);
    device::accel_dist_set_kernel<<<blocks, MISC_CUDA_THREADS>>>(
            grid.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            tmp.data_ptr<bool>());

    device::accel_dist_prop_kernel<<<blocks, MISC_CUDA_THREADS>>>(
            grid.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            tmp.data_ptr<bool>());


    // int32_t* tmp;
    // const int req_size = sz_x * sz_y * sz_z;
    // cuda(Malloc(&tmp, req_size * sizeof(int32_t)));
    // cuda(MemsetAsync(tmp, 0, req_size * sizeof(int32_t)));
    //
    // {
    //     for (int d2 = 0; d2 < 3; ++d2) {
    //         int d0 = d2 == 0 ? 1 : 0;
    //         int d1 = 3 - d0 - d2;
    //         int Q = grid.size(d0) * grid.size(d1);
    //
    //         const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);
    //
    //         device::accel_linf_dist_transform_kernel<<<blocks, MISC_CUDA_THREADS>>>(
    //                 grid.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
    //                 tmp,
    //                 d2);
    //     }
    // }

    // cuda(Free(tmp));
    CUDA_CHECK_ERRORS;
}

void grid_weight_render(
    torch::Tensor data, CameraSpec& cam,
    float step_size,
    float stop_thresh,
    bool last_sample_opaque,
    torch::Tensor offset, torch::Tensor scaling,
    torch::Tensor grid_weight_out) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(offset);
    CHECK_INPUT(scaling);
    CHECK_INPUT(grid_weight_out);
    cam.check();
    const size_t Q = size_t(cam.width) * cam.height;

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    device::grid_weight_render_kernel<<<blocks, MISC_CUDA_THREADS>>>(
        data.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        cam,
        step_size,
        stop_thresh,
        last_sample_opaque,
        offset.data_ptr<float>(),
        scaling.data_ptr<float>(),
        grid_weight_out.packed_accessor32<float, 3, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
}

// For debugging
// void sample_cubemap(torch::Tensor cubemap, // (6, R, R, C)
//                     torch::Tensor dirs,
//                     bool eac,
//                     // Output
//                     torch::Tensor result) {
//     DEVICE_GUARD(cubemap);
//     CHECK_INPUT(cubemap);
//     CHECK_INPUT(dirs);
//     CHECK_INPUT(result);
//     TORCH_CHECK(cubemap.ndimension() == 4);
//     TORCH_CHECK(cubemap.size(0) == 6);
//     TORCH_CHECK(cubemap.size(1) == cubemap.size(2));
//
//     const size_t Q = size_t(dirs.size(0)) * cubemap.size(3);
//     const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);
//
//     device::sample_cubemap_kernel<<<blocks, MISC_CUDA_THREADS>>>(
//         cubemap.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
//         dirs.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
//         Q,
//         eac,
//         // Output
//         result.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
//     CUDA_CHECK_ERRORS;
// }
