#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// nearest-neighbor baseline
// Background is NOT supported
#include <torch/extension.h>
#include <cstdint>
#include "cuda_util.cuh"
#include "render_util.cuh"
#include "data_spec_packed.cuh"

namespace {
namespace device {
// From old version (name is hacky whatever)
struct BasicSingleRaySpec {
    __device__ BasicSingleRaySpec(const float* __restrict__ origin, const float* __restrict__ dir)
        : origin{origin[0], origin[1], origin[2]},
        dir{dir[0], dir[1], dir[2]},
        vdir(dir) {}
    float origin[3];
    float dir[3];
    const float* __restrict__ vdir;
};

__device__ __inline__ float compute_skip_dist_nn(
        const BasicSingleRaySpec& __restrict__ ray,
        const float* __restrict__ invdir,
        const float* __restrict__ pos,
        const int32_t* __restrict__ l,
        int32_t link_val) {
    const uint32_t dist = -link_val;
    const uint32_t cell_ul_shift = (dist - 1);
    const uint32_t cell_side_len = (1 << cell_ul_shift);

    // AABB intersection
    // Consider caching the invdir for the ray
    float tmax = 1e9f;
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        int ul = (((l[i]) >> cell_ul_shift) << cell_ul_shift);
        ul -= l[i];

        const float t1 = (ul - pos[i]) * invdir[i];
        const float t2 = (ul + cell_side_len - pos[i]) * invdir[i];
        if (ray.dir[i] != 0.f) {
            tmax = fminf(tmax, fmaxf(t1, t2));
        }
    }
    return tmax;
}

__device__ __inline__ void trace_ray(
        const PackedSparseGridSpec& __restrict__ grid,
        BasicSingleRaySpec ray,
        RenderOptions& __restrict__ opt,
        float* __restrict__ out) {
    // Warning: modifies ray.origin
    transform_coord(ray.origin, grid._scaling, grid._offset);
    // Warning: modifies ray.dir
    const float delta_scale = _get_delta_scale(grid._scaling, ray.dir);

    float t, tmax;
    float invdir[3];

#pragma unroll 3
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / ray.dir[i];
        if (ray.dir[i] == 0.f)
            invdir[i] = 1e9f;
    }

    {
        float t1, t2;
        t = 0.0f;
        tmax = 1e9f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            ray.origin[i] += 0.5f;  // Fix offset of nn vs lerp
            t1 = (0.0f - ray.origin[i]) * invdir[i];
            t2 = (grid.size[i] - 1.f - ray.origin[i]) * invdir[i];
            t = fmaxf(t, fminf(t1, t2));
            tmax = fminf(tmax, fmaxf(t1, t2));
        }
    }

    if (t > tmax) {
        // Ray doesn't hit box
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            out[j] = opt.background_brightness;
        }
        return;
    } else {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            out[j] = 0.f;
        }
        float pos[3];
        int32_t l[3];
        float basis_fn[9];
        // vdir is unscaled unit dir in world space, for calculating spherical function
        calc_sh(grid.basis_dim, ray.vdir, basis_fn);

        float log_transmittance = 0.f;
        while (t < tmax) {
#pragma unroll 3
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
                pos[j] = fminf(fmaxf(pos[j], 0.f), grid.size[j] - 1.f);
                l[j] = min(static_cast<int32_t>(pos[j]), grid.size[j] - 1);
                pos[j] -= l[j];
            }

            const int32_t link = grid.links[
                (l[0] * grid.size[1] +  l[1]) * grid.size[2] + l[2]
            ];
            if (link >= 0) {
                const float delta_t = _intersect_aabb_unit(pos, invdir) + 1e-2f;
                t += delta_t;
                float sigma = grid.density_data[link];
                if (opt.last_sample_opaque && t + opt.step_size > tmax) {
                    sigma += 1e9;
                }
                if (sigma > opt.sigma_thresh) {
                    const float* __restrict__ sample_val = &grid.sh_data[size_t(link) * grid.sh_data_dim];
                    const float log_transmit = -delta_t * delta_scale * sigma;
                    const float transmittance = expf(log_transmittance);
                    const float weight = transmittance * (1.f - expf(log_transmit));
#pragma unroll 3
                    for (int k = 0; k < 3; ++k) {
                        const int off = k * grid.basis_dim;
                        float tmp = 0.5f;
                        for (int i = 0; i < grid.basis_dim; ++i) {
                            tmp += basis_fn[i] * sample_val[off + i];
                        }
                        out[k] += weight * fmaxf(tmp, 0.f);
                    }
                    log_transmittance += log_transmit;

                    if (transmittance <= opt.stop_thresh) {
                        // Full opacity, stop
                        float scale = 1.0 / (1.0 - transmittance);
                        for (int j = 0; j < 3; ++j) {
                            out[j] *= scale;
                        }
                        return;
                    }
                }
            } else {
                float skip = fmaxf(compute_skip_dist_nn(ray,
                       invdir,
                       pos,
                       l, link), 0.f);
                t += skip + 1e-2f;
            }
        }
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            out[j] += expf(log_transmittance) * opt.background_brightness;
        }
    }
}

__device__ __inline__ void trace_ray_backward(
    const PackedSparseGridSpec& __restrict__ grid,
    const float* __restrict__ grad_output,
    const float* __restrict__ color_cache,
        BasicSingleRaySpec ray,
        RenderOptions& __restrict__ opt,
    PackedGridOutputGrads& __restrict__ grads) {
    // Warning: modifies ray.origin
    transform_coord(ray.origin, grid._scaling, grid._offset);
    // Warning: modifies ray.dir
    const float delta_scale = _get_delta_scale(grid._scaling, ray.dir);

    float t, tmax;
    float invdir[3];

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / ray.dir[i];
        if (ray.dir[i] == 0.0f) {
            invdir[i] = 1e9f;
        }
    }
    {
        float t1, t2;
        t = 0.0f;
        tmax = 1e9f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            ray.origin[i] += 0.5f;  // Fix offset of nn vs lerp
            t1 = (0.0f - ray.origin[i]) * invdir[i];
            t2 = (grid.size[i] - 1.f - ray.origin[i]) * invdir[i];
            if (ray.dir[i] != 0.0f) {
                t = fmaxf(t, fminf(t1, t2));
                tmax = fminf(tmax, fmaxf(t1, t2));
            }
        }
    }

    if (t > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        float pos[3];
        int32_t l[3];
        float basis_fn[9];
        calc_sh(grid.basis_dim, ray.vdir, basis_fn);

        float accum = color_cache[0] * grad_output[0] +
            color_cache[1] * grad_output[1] +
            color_cache[2] * grad_output[2];
        float log_transmittance = 0.f;
        while (t < tmax) {
#pragma unroll 3
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
                pos[j] = fminf(fmaxf(pos[j], 0.f), grid.size[j] - 1.f);
                l[j] = min(static_cast<int32_t>(pos[j]), grid.size[j] - 1);
                pos[j] -= l[j];
            }
            const int32_t link = grid.links[
                (l[0] * grid.size[1] +  l[1]) * grid.size[2] + l[2]
            ];
            if (link >= 0) {
                float delta_t = _intersect_aabb_unit(pos, invdir) + 1e-2f;
                t += delta_t;
                float sigma = grid.density_data[link];
                if (opt.last_sample_opaque && t + opt.step_size > tmax) {
                    sigma += 1e9;
                }
                if (sigma > opt.sigma_thresh) {
                    const float* __restrict__ sample_val = &grid.sh_data[size_t(link) * grid.sh_data_dim];
                    float* __restrict__ grad_sample_val = &grads.grad_sh_out[size_t(link) * grid.sh_data_dim];
                    delta_t *= delta_scale;
                    const float log_transmit = -delta_t * sigma;
                    const float weight = expf(log_transmittance) * (1.f - expf(log_transmit));

                    float total_color = 0.f;
#pragma unroll 3
                    for (int k = 0; k < 3; ++ k) {
                        const int off = k * grid.basis_dim;
                        float tmp = 0.5f;
                        for (int i = 0; i < grid.basis_dim; ++i) {
                            tmp += basis_fn[i] * sample_val[off + i];
                        }

                        if (tmp > 0.f) {
                            total_color += tmp * grad_output[k];
                            tmp = weight * grad_output[k];
                            for (int i = 0; i < grid.basis_dim; ++i) {
                                atomicAdd(&grad_sample_val[off + i],
                                        basis_fn[i] * tmp);
                            }
                        }
                    }
                    log_transmittance += log_transmit;
                    accum -= weight * total_color;
                    if (grads.mask_out != nullptr) {
                        grads.mask_out[link] = true;
                    }
                    atomicAdd(&grads.grad_density_out[link],
                            delta_t * (total_color *
                                expf(log_transmittance) - accum));
                    if (expf(log_transmittance) <= opt.stop_thresh) {
                        return;
                    }
                }
            } else {
                t += fmaxf(compute_skip_dist_nn(ray,
                       invdir,
                       pos,
                       l, link), 0.f) + 1e-2f;
            }
        }
    }
}  // trace_ray_backward


// ** Kernels

__global__ void render_ray_svox1_kernel(
        PackedSparseGridSpec grid,
        PackedRaysSpec rays,
        RenderOptions opt,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits>
        out) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    trace_ray(
        grid,
        BasicSingleRaySpec(&rays.origins[tid][0], &rays.dirs[tid][0]),
        opt,
        &out[tid][0]);
}


__global__ void render_ray_svox1_backward_kernel(
    PackedSparseGridSpec grid,
    const float* __restrict__ grad_output,
    const float* __restrict__ color_cache,
        PackedRaysSpec rays,
        RenderOptions opt,
    bool grad_out_is_rgb,
    PackedGridOutputGrads grads
        ) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));

    float grad_out[3];
    if (grad_out_is_rgb) {
        const float norm_factor = 2.f / (3 * int(rays.origins.size(0)));
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float resid = color_cache[tid * 3 + i] - grad_output[tid * 3 + i];
            grad_out[i] = resid * norm_factor;
        }
    } else {
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            grad_out[i] = grad_output[tid * 3 + i];
        }
    }

    trace_ray_backward(
        grid,
        grad_out,
        color_cache + tid * 3,
        BasicSingleRaySpec(&rays.origins[tid][0], &rays.dirs[tid][0]),
        opt,
        grads);
}

}  // namespace device
}  // namespace

torch::Tensor volume_render_svox1(SparseGridSpec& grid, RaysSpec& rays, RenderOptions& opt) {
    DEVICE_GUARD(grid.sh_data);
    TORCH_CHECK(grid.basis_type == BASIS_TYPE_SH); // Only supporting SH for now
    grid.check();
    rays.check();
    const auto Q = rays.origins.size(0);

    const int cuda_n_threads = 512;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor result = torch::zeros({Q, 3}, rays.origins.options());
    device::render_ray_svox1_kernel<<<blocks, cuda_n_threads>>>(
            grid, rays, opt,
            result.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
    return result;
}

void volume_render_svox1_backward(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor grad_out,
        torch::Tensor color_cache,
        GridOutputGrads& grads) {
    DEVICE_GUARD(grid.sh_data);
    grid.check();
    rays.check();
    grads.check();
    CHECK_INPUT(grad_out);
    CHECK_INPUT(color_cache);

    const int Q = rays.origins.size(0);

    const int cuda_n_threads = 512;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    device::render_ray_svox1_backward_kernel<<<blocks, cuda_n_threads>>>(
        grid,
        grad_out.data_ptr<float>(),
        color_cache.data_ptr<float>(),
        rays,
        opt,
        false,
        grads);
    CUDA_CHECK_ERRORS;
}

void volume_render_svox1_fused(
        SparseGridSpec& grid,
        RaysSpec& rays,
        RenderOptions& opt,
        torch::Tensor rgb_gt,
        float _,  // not supported
        float _2, // not supported
        torch::Tensor rgb_out,
        GridOutputGrads& grads) {

    DEVICE_GUARD(grid.sh_data);
    CHECK_INPUT(rgb_gt);
    CHECK_INPUT(rgb_out);
    grid.check();
    rays.check();
    grads.check();
    const auto Q = rays.origins.size(0);

    const int cuda_n_threads = 512;
    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
        device::render_ray_svox1_kernel<<<blocks, cuda_n_threads>>>(
                grid, rays, opt,
                // Output
                rgb_out.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
        CUDA_CHECK_ERRORS;
    }
    {
        const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
        device::render_ray_svox1_backward_kernel<<<blocks, cuda_n_threads>>>(
                grid,
                rgb_gt.data_ptr<float>(),
                rgb_out.data_ptr<float>(),
                rays, opt,
                true,
                // Output
                grads);
        CUDA_CHECK_ERRORS;
    }
}
